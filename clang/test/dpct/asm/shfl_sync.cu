// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/shfl_sync %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/shfl_sync/shfl_sync.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/shfl_sync/shfl_sync.dp.cpp -o %T/shfl_sync/shfl_sync.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void shfl_sync() {
    int value; 
    unsigned mask = 0xFFFFFFFF;
    int offset;
    int output;                                              

    // CHECK:    /*
    // CHECK-NEXT:    DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for shift_sub_group_right. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate inline PTX asm instruction shfl.sync.up.b32.
    // CHECK-NEXT:    */
    // CHECK-NEXT:    output = dpct::shift_sub_group_right(item_ct1.get_sub_group(), value, offset);
    asm volatile("shfl.sync.up.b32 %0, %1, %2, %3, %4;" : "=r"(output) : "r"(value), "r"(offset), "r"(0), "r"(mask));

    // CHECK:    /*
    // CHECK-NEXT:    DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for shift_sub_group_left. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate inline PTX asm instruction shfl.sync.down.b32.
    // CHECK-NEXT:    */
    // CHECK-NEXT    output = dpct::shift_sub_group_left(item_ct1.get_sub_group(), value, offset);  
    asm volatile("shfl.sync.down.b32 %0, %1, %2, %3, %4;" : "=r"(output) : "r"(value), "r"(offset), "r"(0), "r"(mask));

    // CHECK:    /*
    // CHECK-NEXT:    DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for select_from_sub_group. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate inline PTX asm instruction shfl.sync.idx.b32.
    // CHECK-NEXT:    */ 
    // CHECK-NEXT    output = dpct::select_from_sub_group(item_ct1.get_sub_group(), value, offset);  
    asm volatile("shfl.sync.idx.b32 %0, %1, %2, %3, %4;" : "=r"(output) : "r"(value), "r"(offset), "r"(0), "r"(mask));

    // CHECK:    /*
    // CHECK-NEXT:    DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate inline PTX asm instruction shfl.sync.bfly.b32.
    // CHECK-NEXT:    */
    // CHECK-NEXT:    output = dpct::permute_sub_group_by_xor(item_ct1.get_sub_group(), value, offset); 
    asm volatile("shfl.sync.bfly.b32 %0, %1, %2, %3, %4;" : "=r"(output) : "r"(value), "r"(offset), "r"(0), "r"(mask));
}

// clang-format off
