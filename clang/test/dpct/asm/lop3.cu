// RUN: dpct -out-root %T/lop3 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/lop3/lop3.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/lop3/lop3.dp.cpp -o %T/lop3/lop3.dp.o %}
// clang-format off

#include <hip/hip_runtime.h>
#include <cstdint>

// a^b^c
static __device__ __forceinline__ uint32_t LOP3LUT_XOR(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d1;
  // CHECK: d1 = dpct::lop3(a, b, c, 0x96);
  asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(d1) : "r"(a), "r"(b), "r"(c));
  return d1;
}

// (a ^ (c & (b ^ a)))
static __device__ __forceinline__ uint32_t LOP3LUT_XORAND(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d2;
  // CHECK: d2 = dpct::lop3(a, c, b, 0xb8);
  asm("lop3.b32 %0, %1, %3, %2, 0xb8;" : "=r"(d2) : "r"(a), "r"(b), "r"(c));
  return d2;
}

// ((a & (b | b)) | (b & b))
static __device__ __forceinline__ uint32_t LOP3LUT_ANDOR(uint32_t a, uint32_t b) {
  uint32_t d3;
  // CHECK: d3 = dpct::lop3(a, b, b, 0xe8);
  asm("lop3.b32 %0, %1, %2, %2, 0xe8;" : "=r"(d3) : "r"(a), "r"(b));
  return d3;
}

#define B 3
__device__  int hard(int a) {
  int d4;
  // CHECK: d4 = dpct::lop3((a + B), B, 3, 0x1C);
  asm("lop3.b32 %0, %1, %2, 3, 0x1C;" : "=r"(d4) : "r"(a + B), "r"(B));
  return d4;
}

// CHECK: template <int lut, typename T> inline T lop3(T a, T b, T c) {
// CHECK-NEXT:  T res;
// CHECK-NEXT:  res = dpct::lop3(a, b, c, lut);
// CHECK-NEXT:  return res;
// CHECK-NEXT:}
template <int lut, typename T> __device__ inline T lop3(T a, T b, T c) {
  T res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(res)
               : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}
// clang-format on
