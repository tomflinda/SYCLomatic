// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/cp %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cp/cp.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/cp/cp.dp.cpp -o %T/cp/cp.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

// CHECK:inline void cp_async_commit_group() {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1026:{{[0-9]+}}: The call to "cp.async.commit_group;" was removed because there is no equivalent functionality in SYCL side. You may need to adjust the code.
// CHECK-NEXT:  */
// CHECK-EMPTY:
// CHECK-NEXT:}
__device__ inline void cp_async_commit_group() {
asm volatile("cp.async.commit_group;" ::);
}


// CHECK:inline void cp_async_wait_group() {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1026:{{[0-9]+}}: The call to "cp.async.wait_group 0;" was removed because there is no equivalent functionality in SYCL side. You may need to adjust the code.
// CHECK-NEXT:  */
// CHECK-EMPTY:
// CHECK-NEXT:}
__device__ inline void cp_async_wait_group() {
asm volatile("cp.async.wait_group 0;");
}

// CHECK:inline void cp_async_wait_all() {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1026:{{[0-9]+}}: The call to "cp.async.wait_all;" was removed because there is no equivalent functionality in SYCL side. You may need to adjust the code.
// CHECK-NEXT:  */
// CHECK-EMPTY:
// CHECK-NEXT:}
__device__ inline void cp_async_wait_all() {
asm volatile("cp.async.wait_all;");
}

// clang-format on
