// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/red %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/red/red.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/red/red.dp.cpp -o %T/red/red.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>

// CHECK: void atomicAddKernel(int* lock, int val) {
// CHECK-NEXT:    *lock += val;
// CHECK-NEXT:}
__global__ void atomicAddKernel(int* lock, int val) {
    asm volatile("red.relaxed.gpu.global.add.s32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// clang-format on
