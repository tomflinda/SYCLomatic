// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/membar_no_exp %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/membar_no_exp/membar_no_exp.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/membar_no_exp/membar_no_exp.dp.cpp -o %T/membar_no_exp/membar_no_exp.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void membar() {

  // CHECK: sycl::group_barrier(item_ct1.get_group());
  asm volatile("membar.cta;":::"memory"); 

  // CHECK:/*
  // CHECK-NEXT: DPCT1087:0: SYCL currently does not support cross group synchronization. You can specify "--use-experimental-features=root-group" to use the root-group to migrate membar.gl;.
  // CHECK-NEXT: */
  asm volatile("membar.gl;":::"memory"); 

  // CHECK:/*
  // CHECK-NEXT:DPCT1053:{{[0-9]+}}: Migration of device assembly code is not supported.
  // CHECK-NEXT:*/
  asm volatile("membar.sys;":::"memory");
}

// clang-format off
