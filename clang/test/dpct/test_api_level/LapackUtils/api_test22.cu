// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test22_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test22_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test22_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test22_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test22_out

// CHECK: 31
// TEST_FEATURE: LapackUtils_syhegvd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipblasFillMode_t uplo;
  int n;
  float *A;
  int lda;
  float *B;
  int ldb;
  float *W;
  float *work;
  int lwork;
  int *info;
  hipsolverSyevjInfo_t params;

  hipsolverDnSsygvj(handle, itype, jobz, uplo, n, A, lda, B, ldb, W, work, lwork,
                   info, params);
  return 0;
}
